#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <algorithm>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

// Cali Regions
const char* main_region = "main";
const char* data_init = "data_init";
const char* comp = "comp";
const char* comm = "comm";
const char* comp_small = "comp_small";
const char* comm_small = "comm_small";
const char* comp_large = "comp_large";
const char* comm_large = "comm_large";
const char* correctness_check = "correctness_check";

using namespace std;

int correctnessCheck(int *arr, int size) {
  CALI_MARK_BEGIN(correctness_check);
  for (int i=0; i<size-1; i++) {
    if (arr[i+1] < arr[i])
      return 0;  // means it's not ordered correctly
  }
  CALI_MARK_END(correctness_check);

  return 1;
}

void dataInit(int *arr, int size, int inputType) {
  CALI_MARK_BEGIN(data_init);
  int numToSwitch = size / 100;
  int firstIndex, secondIndex;
  switch (inputType) {
    case 1:
      // sorted
      for (int i=0; i<size; i++) {
        arr[i] = i;
      }
      break;
    case 2:
      // reverse sorted
      for (int i=0; i<size; i++) {
        arr[i] = size-i;
      }
      break;
    case 3:
      // randomized
      for (int i=0; i<size; i++) {
        arr[i] = rand() % RAND_MAX;
      }
      break;
    case 4:
      // 1% perturbed
      for (int i=0; i<size; i++) {
        arr[i] = i;
      }
      if (numToSwitch == 0)  // at the very least one value should be switched
        numToSwitch = 1;
      
      for (int i=0; i<numToSwitch; i++) {
        firstIndex = rand() % size;
        secondIndex = rand() % size;
        //printf("first index: %d, second index: %d\n", firstIndex, secondIndex);
        while (firstIndex == secondIndex) {
          secondIndex = rand() % size;
        } 
        std::swap(arr[firstIndex], arr[secondIndex]); 
      }
      break;
    default:
      printf("THAT'S NOT A VALID INPUT TYPE");
      break;
  }
  
  CALI_MARK_END(data_init);
}

void finalSort(int** buckets, int rows) {
    printf("entered final sort\n");
    for (int r = 0; r < rows; ++r) {
        std::sort(buckets[r], buckets[r] + NUM_VALS);
    }
}

void chooseSplitters(int *splitters, int *samples) {
    cout << "entered choose splitters" << endl;
    // sort samples
    cout << "BLOCKS: " << BLOCKS << endl;
    int samplesSize = 5 * BLOCKS;
    cout << "samplesSize: " << samplesSize << endl;
    std::sort(samples, samples + samplesSize);
    
    // choose splitters
    int spacing = std::ceil((float)samplesSize/(float)THREADS);
    int splitterIndex = spacing-1;
    
    for (int i = 0; i < THREADS; i++) {
      if (splitterIndex > samplesSize-1) {
        cout << "BROKE CHOOSE SPLITTERS EARLY AT i= " << i << endl;
        break;     
      }
      // cout << "i: " << i << "index: " << splitterIndex << " samples[index]: " << samples[splitterIndex] << endl; 
      splitters[i] = samples[splitterIndex];
      splitterIndex += spacing;
    }
}


__global__ void chooseSamples(int* data, int *samples, int numBlocks) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // only smallest thread sorts block
    if (threadIdx.x == 0) {
      // sort each block
      //printf("choose samples data[%d]: %d\n", index, data[index]);
      for (int i = 0; i < blockDim.x - 1; ++i) {
        for (int j = 0; j < blockDim.x - i - 1; ++j) {
            if (data[index + j] > data[index + j + 1]) {
                // Swap elements if they are in the wrong order
                int temp = data[index + j];
                data[index + j] = data[index + j + 1];
                data[index + j + 1] = temp;
            }
        }
      }
      
      // choose samples from sorted block
      int spacing = blockDim.x / 5;
      int sampleIndex = spacing-1;
      
      for (int i = 0; i < 5; i++) {
        if (index+sampleIndex > blockDim.x * numBlocks) {
          printf("BROKE CHOOSE SAMPLES EARLY AT i=%d\n", i);
          break;
        }
        samples[blockIdx.x * 5 + i] = data[index+sampleIndex];
        // printf("index: %d, chosen sample: %d\n", blockIdx.x * (numBlocks-1) +i, data[index+sampleIndex]);
        sampleIndex += spacing;
      }
    }

    
}

__global__ void sampleSort(int* data, int** buckets, int* splitters, int* flattenedArr, int numSplitters, int numVals) {
    
    // printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index == 0)
      printf("numSplitters: %d", numSplitters);
      
    // each thread checks which bucket they fall into
    int j = 0;
    //printf("index: %d \n", index, j);
    while(j < numSplitters) {  // j being which bucket it should belong to
  			if (j == numSplitters-1) {
          // printf("data[%d]: %d went into buckets[%d][%d]\n", index, data[index], j, index);
          // means it should go in last bucket
          // makes sure that we don't try to access splitters[buckets.size()-1]. will go out of range
          buckets[j][index] = data[index];
          break;
        }
        if(data[index] < splitters[j]) {
          // printf("data[%d]: %d went into buckets[%d][%d]\n", index, data[index], j, index);  
          __syncthreads();      
  				buckets[j][index] = data[index];
          break;
  			}
  			j++;
    }
    
    __syncthreads();
    
    // store bucket values in flattened array. Only have one thread do it
    if (index == 0) {
      printf("entering bucket values\n");
      int arrIndex = 0;
      for (int i = 0; i < numSplitters; ++i) {
          for (int j = 0; j < numVals; ++j) {
              // printf("arrIndex: %d, value: %d\n", arrIndex, buckets[i][j]);
              flattenedArr[arrIndex++] = buckets[i][j];
          }
      }
    }
} 
  
int main(int argc, char *argv[])
{
    int inputType;
    inputType = atoi(argv[3]);
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);
    printf("Input type: %d\n", inputType);
    
    int device = 0;  // Assuming device 0, change if needed

    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    std::cout << "Total Global Memory: " << prop.totalGlobalMem << " bytes" << std::endl;
    
    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();
    
    CALI_MARK_BEGIN(main_region);

    // host data
    int* hostData = new int[NUM_VALS];
    int *splitters = new int[THREADS]; 
    int *samples = (int*)malloc(sizeof(int) * 5*BLOCKS);  // each block picks out 5 potential splitter candidates
    
    // initialize data according to inputType
    dataInit(hostData, NUM_VALS, inputType);
    
    /*
    cout << "original arr" << endl;  
    for (int i = 0; i < NUM_VALS; ++i) {
        cout << hostData[i] << " ";
    }
    cout << endl;  
    */

    // device data
    int* devData, *dsplitters, *dsamples;
    hipMalloc((void**)&devData, NUM_VALS * sizeof(int));
    hipMalloc((void**)&dsplitters, THREADS * sizeof(int));
    hipMalloc((void**)&dsamples, 5*BLOCKS * sizeof(int));
    cout << "device data" << endl;
    
    // send chunks to device
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    hipMemcpy(devData, hostData, NUM_VALS * sizeof(int), hipMemcpyHostToDevice);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);
    cout << "send chunks" << endl;
    
    // have device sort and send back samples
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    chooseSamples<<<BLOCKS, THREADS>>>(devData, dsamples, BLOCKS);
    hipDeviceSynchronize();
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);
    cout << "choose samples" << endl;
    
    // receive samples from device
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_small);
    hipMemcpy(samples, dsamples, 5 * BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
    CALI_MARK_END(comm_small);
    CALI_MARK_END(comm);
    cout << "receive samples" << endl;
    hipFree(dsamples);
    /*
    for (int i=0; i<5*BLOCKS; i++) {
      cout << samples[i] << " ";
    }
    cout << endl;
    */
    
    // sort samples and choose splitters
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_small);
    chooseSplitters(splitters, samples);
    CALI_MARK_END(comp_small);
    CALI_MARK_END(comp);
    cout << "choose splitters" << endl;
    
    
    for (int i=0; i<THREADS; i++) {
      cout << splitters[i] << " ";
    }
    cout << endl;
    
    
    // allocate memory for host and device 2d bucket arrays
    int rows = THREADS;
    int** buckets = new int*[rows];
    int** dbuckets;
    int* dflattenedArr;
    hipMalloc((void**)&dflattenedArr, rows * NUM_VALS * sizeof(int));
    for (int i = 0; i < rows; ++i) {
        buckets[i] = new int[NUM_VALS];
    }
    
    // initalize buckets with -1 so we know what to remove later
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < NUM_VALS; j++) {
        buckets[i][j] = -1;
      }
    }
    
    // Allocate device memory for the 2D array
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    hipMalloc((void**)&dbuckets, rows * sizeof(int*));
    for (int i = 0; i < rows; ++i) {
        int* d_row;
        hipMalloc((void**)&d_row, NUM_VALS * sizeof(int));
        hipMemcpy(d_row, buckets[i], NUM_VALS * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dbuckets + i, &d_row, sizeof(int*), hipMemcpyHostToDevice);
    }
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);
    
    // send chunks to device w/ splitters
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_small);
    hipMemcpy(dsplitters, splitters, sizeof(int) * THREADS, hipMemcpyHostToDevice);
    CALI_MARK_END(comm_small);
    CALI_MARK_END(comm);
    cout << "send chunks" << endl;
    
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    sampleSort<<<BLOCKS, THREADS>>>(devData, dbuckets, dsplitters, dflattenedArr, THREADS, NUM_VALS);
    hipDeviceSynchronize();
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        // Handle the error appropriately
    }
    printf("cudaErr: \n", cudaErr);
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);
    cout << "sample sort" << endl;
    
    cout << "before malloc" << endl;
    // int *flattenedArr = (int*)malloc(sizeof(int) * (BLOCKS-1)*NUM_VALS);
    int *flattenedArr;
    hipHostMalloc((void**)&flattenedArr, THREADS * NUM_VALS * sizeof(int));
    cout << "after malloc" << endl;
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    cout << "before hipMemcpy" << endl;
    hipMemcpy(flattenedArr, dflattenedArr, THREADS * NUM_VALS * sizeof(int), hipMemcpyDeviceToHost);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);
    
    cout << "before initialize" << endl;
    
    // initializing unflattened arr
    int** unflattened = new int*[rows];
    for (int i = 0; i < rows; ++i) {
        unflattened[i] = new int[NUM_VALS];
    }
    
    /*
    for (int i = 0; i < THREADS; i++) {
      cout << "i: " << i << endl;
      cout << flattenedArr[i] << endl;
    }
    */
    
    
    cout << "before unflatten" << endl;
    
    // unflatten the arr
    int index = 0;
    for (int i = 0; i < rows; ++i) {
        // cout << "ROW: " << i << endl;
        for (int j = 0; j < NUM_VALS; ++j) {
            //cout << "j: " << j << "index: " << index << endl;
            unflattened[i][j] = flattenedArr[index++];
        }
    }
    
    cout << "finished unflattened" << endl;
    
    // final sort each row
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    finalSort(unflattened, rows);
    cout << "final sort" << endl;
    
    // append to one array and done!
    int* finalArr = new int[NUM_VALS];
    int finalArrIndex = 0;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; ++j) {
            if (unflattened[i][j] != -1) {
              finalArr[finalArrIndex++] = unflattened[i][j];
            }
        }
        
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);
    
    cout << "FINAL ARRAY" << endl;
    for (int i = 0; i < NUM_VALS; i++) {
      cout << finalArr[i] << " ";
    }
    
    if (correctnessCheck(finalArr, NUM_VALS)) {
      printf("\nCORRECT");
    } else {
      printf("\nINCORRECT");
    }
    
    
    CALI_MARK_END(main_region);
    
    const char* algorithm = "Sample Sort";
    const char* programmingModel = "CUDA";
    const char* datatype = "int";
    const char* inputTypeStr;
    switch (inputType) {
      case 1:
        inputTypeStr = "Sorted";
        break;
      case 2:
        inputTypeStr = "Reverse Sorted";
        break;
      case 3:
        inputTypeStr = "Random";
        break;
      case 4:
        inputTypeStr = "1% Perturbed";
        break;
      default:
        inputTypeStr = "No input type. Invalid input argument entered";
        break;
    }
    
    adiak::init(NULL);
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();   
    adiak::clustername();  
    adiak::value("Algorithm", algorithm);
    adiak::value("ProgrammingModel", programmingModel); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", datatype); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", 4); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", inputTypeStr); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", THREADS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    adiak::value("group_num", 10); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Handwritten");
  
    adiak::value("main", main_region);
    adiak::value("data_init", data_init);
    adiak::value("comm", comm);
    adiak::value("comp", comp);
    adiak::value("comm_large", comm_large);
    adiak::value("comm_small", comm_small);
    adiak::value("comp_large", comp_large);
    adiak::value("comp_small", comp_small);
    adiak::value("correctness_check", correctness_check);

    // Flush Caliper output before finalizing MPI
    mgr.stop();
    mgr.flush();
};

